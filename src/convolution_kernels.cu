#include "hip/hip_runtime.h"
#include "float.h"
#include "limits.h"
#define CEILING(x) (int)(x) + (1 - (int)((int)((x) + 1) - (x)))

__global__ void convolve_sequence(const {{ data_type }} *input,
				  {{ data_type }} *target, const {{ data_type }} *filter,
				  const {{ data_type }} *bias, const unsigned int width, 
				  const unsigned int height, const unsigned int filter_width, 
				  const unsigned int n_filters, const unsigned int stride) {

  /* Performs a 1D convolution on each row of a matrix with 
     multiple filters. Filter size must be even and input is
     padded on the right with zeros.
  */
    
  const unsigned int i = blockIdx.y;
  const unsigned int j = blockIdx.x*blockDim.x+threadIdx.x;
  const unsigned int f = blockIdx.z;
  const unsigned int lin_idx = i*width+j;
  const unsigned int row_start = i*width;
  const unsigned int target_width = CEILING((double) width / stride);
  unsigned int shared_idx, input_idx;    
    
  const unsigned int shared_width = blockDim.x+filter_width-1;
  extern __shared__ {{ data_type }} sdata[];
  {{ data_type }} *input_shared = sdata;
  {{ data_type }} *bias_shared = input_shared + shared_width;
    
  const unsigned int halo_width = filter_width - 1;
    
  shared_idx = threadIdx.x;
  input_shared[shared_idx] = (j < width && i < height) ? input[lin_idx] : 0;
  __syncthreads();

  if (i < height) {
    int halo_index_right = (blockIdx.x+1)*blockDim.x+threadIdx.x;
    if (threadIdx.x < halo_width) {
      shared_idx = blockDim.x+threadIdx.x;
      input_idx = row_start+halo_index_right;
      input_shared[shared_idx] =
	(halo_index_right >= width) ? 0 : input[input_idx];
    }
  }

  if (threadIdx.x < n_filters)
    bias_shared[threadIdx.x] = bias[threadIdx.x];
  __syncthreads();
  
  unsigned int filter_idx, target_idx;
  if (!(j%stride) && i < height && j < width) {
    {{ data_type }} Pvalue = bias_shared[f];
    for (int k=0; k < filter_width; k++) {
      shared_idx = threadIdx.x+k;
      filter_idx = f*filter_width+k;
      Pvalue += input_shared[shared_idx]*filter[filter_idx];
    }
    target_idx = i*n_filters*target_width+f*target_width+j/stride;
    target[target_idx] = Pvalue;
  }
}

__global__ void gradient_reduce(const {{ data_type }} *df_weights,
    {{ data_type }} *df_weights_sum, const unsigned int n_filters,
    const unsigned int filter_width, const unsigned int n_elements) {

    /* Completes the reduction operation of conv1d_grad_weight
    */
    
    const unsigned int tid = threadIdx.x;
    const unsigned int df_weights_idx = blockIdx.x*filter_width*n_elements+
        blockIdx.y*n_elements+threadIdx.x;
    
    extern __shared__ {{ data_type }} sdata[];
    
    sdata[tid] = (tid<n_elements) ? df_weights[df_weights_idx] : 0;
    if (tid+blockDim.x < n_elements)
        sdata[tid] += df_weights[df_weights_idx+blockDim.x];
    __syncthreads();
    
    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (tid < s) {
            sdata[tid] += sdata[tid+s];
        }
        __syncthreads();
    }
    
    if (tid==0) {        
        const unsigned int df_weights_sum_idx = blockIdx.x*filter_width+blockIdx.y;
        df_weights_sum[df_weights_sum_idx] = sdata[0];
    }
}

__global__ void convolve_sequence_gradient(
					   const {{ data_type }} *input, const {{ data_type }} *df_output,
					   {{ data_type }} *df_weights, const unsigned int width,
					   const unsigned int height, const unsigned int filter_width,
					   const unsigned int n_filters) {
  
  const unsigned int stride = 4;
  const unsigned int tx = threadIdx.x;
  const unsigned int f = blockIdx.y*blockDim.y+threadIdx.y;
  const unsigned int input_idx = blockIdx.x*blockDim.x+tx;
  const unsigned int column = input_idx % width;
  const unsigned int column_start_block = (blockIdx.x*blockDim.x)%width; // Column of first thread in block
  const unsigned int row_start_block = (blockIdx.x*blockDim.x)/width; // Row of first thread in block
  const unsigned int len_input = height*width;
  const unsigned int output_width = width / stride;

  unsigned int df_weights_idx, output_idx, shared_idx, df_output_offset, row_shared, column_shared;
  int halo_idx;
  
  const unsigned int halo_width = (filter_width / stride) - 1;
  const unsigned int shared_width = halo_width + blockDim.x / stride;
  extern __shared__ {{ data_type }} sdata[];
  {{ data_type }} *df_output_shared = sdata;
  {{ data_type }} *df_weights_reduce = df_output_shared + shared_width;

  const {{ data_type }} input_element = 
			  (input_idx < len_input) ? input[input_idx] : 0.;

  // Load halo elements on the left
  if (tx < halo_width) {
    output_idx = row_start_block*n_filters*output_width+
      f*output_width+column_start_block/stride-halo_width+tx;
    shared_idx = tx;
    halo_idx = column_start_block / stride - halo_width + tx;
    df_output_shared[shared_idx] = 
      (halo_idx < 0) ? 0. : df_output[output_idx];
  }

  if (tx < blockDim.x/stride) {
    column_shared = ((blockIdx.x*blockDim.x % width) / stride + tx) % output_width;
    row_shared = (blockIdx.x*blockDim.x+stride*tx) / width;
    output_idx = row_shared*n_filters*output_width+f*output_width+column_shared;
    df_output_shared[tx+halo_width] = 
      (column_shared < output_width && row_shared < height) ?
      df_output[output_idx] : 0.;
  }
  
  __syncthreads();

  for (unsigned int k=0; k<(halo_width+1); k++) {
    df_output_offset = (halo_width-k)*stride;
    df_weights_reduce[tx] =
      (column >= df_output_offset)
      ? input_element * df_output_shared[tx/stride+k] : 0.;
    __syncthreads();

    for (unsigned int s=blockDim.x/2; s>=stride; s>>=1) {
      if (tx<s) {
	df_weights_reduce[tx] += df_weights_reduce[tx+s];
      }
      __syncthreads();
    }

    if (tx<stride) {
      df_weights_idx = 
	f*filter_width*gridDim.x+
	(tx+df_output_offset)*gridDim.x+
	blockIdx.x;
      df_weights[df_weights_idx] = df_weights_reduce[tx];
    }
  }
}

__global__ void max_pool(const {{ data_type }} *mat,
    {{ data_type }} *target, 
    unsigned int *argmax,
    const unsigned int height,
    const unsigned int width,
    const unsigned int pool_size) {

    /* Perform 1D max-pooling on all rows of a matrix
    */
    
    const unsigned int tx = threadIdx.x;
    const unsigned int i = blockIdx.y;
    const unsigned int j = blockIdx.x*pool_size+tx;
    const unsigned int mat_idx = blockIdx.z*height*width+i*width+j;
    
    extern __shared__ {{ data_type }} sdata[];
    {{ data_type }} *max_shared = sdata;
    unsigned int *argmax_shared = (unsigned int*) (max_shared + blockDim.x);
    
    max_shared[tx] = (i < height && j < width && tx < pool_size) ? mat[mat_idx] : -FLT_MAX;
    argmax_shared[tx] = (i < height && j < width && tx < pool_size) ? j : UINT_MAX;
    __syncthreads();
    
    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (tx<s && sdata[tx+s] > sdata[tx]) {
            max_shared[tx] = max_shared[tx+s];
	    argmax_shared[tx] = argmax_shared[tx+s];
        }
        __syncthreads();
    }
    
    if (tx==0) {
      const unsigned int target_idx = blockIdx.y*gridDim.z*gridDim.x+
	blockIdx.z*gridDim.x+blockIdx.x;
        target[target_idx] = max_shared[0];
	argmax[target_idx] = argmax_shared[0];
    }
}

__global__ void max_pool_gradient(
    const unsigned int *argmax,
    const {{ data_type }} *df_output,
    {{ data_type }} *df_input,
    const unsigned int height,
    const unsigned int width,
    const unsigned int width_pooled) {

    /* Gradient of max-pooling operation
    */
    
    const unsigned int tx = threadIdx.x;
    const unsigned int bx = blockIdx.x;
    const unsigned int by = blockIdx.y;
    const unsigned int bz = blockIdx.z;
    const unsigned int n_filters = gridDim.y;
    const unsigned int column = bx*blockDim.x+tx;
    
    const unsigned int max_idx = argmax[bz*n_filters*width_pooled+
					by*width_pooled+bx];
    const {{ data_type }} df_output_element = df_output[bz*n_filters*width_pooled+
							by*width_pooled+bx];

    if (bx*blockDim.x+tx < width) {
        df_input[by*height*width+bz*width+bx*blockDim.x+tx] =
            (column == max_idx) ? df_output_element : 0.;
    }
}
