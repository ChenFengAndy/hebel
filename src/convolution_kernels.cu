#include "hip/hip_runtime.h"
// Copyright (C) 2013  Hannes Bretschneider

// This program is free software; you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation; either version 2 of the License, or
// (at your option) any later version.

// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.

// You should have received a copy of the GNU General Public License along
// with this program; if not, write to the Free Software Foundation, Inc.,
// 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.

#include <float.h>
#include <limits.h>
#include "convolution_kernels.h"

__global__ void convolve_sequence(const nucleotide_t *input,
				  {{ data_type }} *target, 
				  const {{ data_type }} *filter,
				  const {{ data_type }} *bias, 
				  const unsigned int input_offset,
				  const unsigned int width,
				  const unsigned int total_width,
				  const unsigned int height, 
				  const unsigned int filter_width, 
				  const unsigned int total_target_width,
				  const unsigned int target_offset,
				  const unsigned int n_filters) {

  /*
    Convolves a set of filters with an input sequence. This function
    can operate on a subset of columns of the input array and output
    the result to a subset of columns of the output array.

    *input : pointer to the input sequence
    *target : pointer to the output array
    *filter : pointer to the filter bank
    *bias : pointer to the array of biases
    input_offset : column offset when working 
      on a subset of the input array
    width : width of input sequence
    total_width: total width/row stride of the input sequence 
      (for operating on a subset of the input)
    height : number of rows in the input array
    filter_width : width of the filters in nucleotides
    total_target_width : total width/row stride of the target array
    target_offset : column offset into the target array
    n_filters : number of filters

  */
    
  const unsigned int f = blockIdx.y;
  const unsigned int lin_idx = blockIdx.x*blockDim.x+threadIdx.x;
  const unsigned int i = lin_idx / width;
  const unsigned int j = lin_idx % width;
  const unsigned int row_start = i*total_width + input_offset;
  const unsigned int filter_elements = STRIDE*filter_width; // Actual number of elements in filter
  const {{ data_type }} bias_filter = bias[f];
  unsigned int shared_idx, input_idx, target_idx;
  nucleotide_t nt;
    
  const unsigned int shared_width = (blockDim.x+filter_width-1);
  extern __shared__ {{ data_type }} sdata[];
  nucleotide_t *input_shared = (nucleotide_t*) sdata;
  {{ data_type }} *filter_shared = sdata + shared_width;
    
  const unsigned int halo_width = filter_width - 1;
    
  // Load input into shared memory
  shared_idx = threadIdx.x;
  input_idx = i*total_width + input_offset + j;
  input_shared[shared_idx] = (i < height) ? input[input_idx] : DNA_N;
  __syncthreads();

  // Load halo elements on right side
  if (i < height) {
    int halo_index_right = (blockIdx.x+1)*blockDim.x+threadIdx.x;
    if (threadIdx.x < halo_width) {
      shared_idx = blockDim.x+threadIdx.x;
      input_idx = row_start+halo_index_right;
      input_shared[shared_idx] =
	(halo_index_right >= width) ? DNA_N : input[input_idx];
    }
  }

  // Load filter elements into shared memory
  if (threadIdx.x < filter_elements)
    filter_shared[threadIdx.x] = filter[f*filter_elements+threadIdx.x];
  __syncthreads();
  
  // Perform convolution
  if (i < height) {
    {{ data_type }} Pvalue = bias_filter;
    for (int k=0; k < filter_width; k++) {
      if (j+k < width) {
	shared_idx = threadIdx.x+k;
	nt = input_shared[shared_idx];
      
	if (CHECK_NT(nt, DNA_A))
	  Pvalue += filter_shared[STRIDE*k];

	if (CHECK_NT(nt, DNA_C))
	  Pvalue += filter_shared[STRIDE*k+1];

	if (CHECK_NT(nt, DNA_G))
	  Pvalue += filter_shared[STRIDE*k+2];

	if (CHECK_NT(nt, DNA_T))
	  Pvalue += filter_shared[STRIDE*k+3];

	if (CHECK_NT(nt, DNA_R)) {
	  Pvalue += .5 * filter_shared[STRIDE*k];
	  Pvalue += .5 * filter_shared[STRIDE*k+2];
	}

	if (CHECK_NT(nt, DNA_Y)) {
	  Pvalue += .5 * filter_shared[STRIDE*k+1];
	  Pvalue += .5 * filter_shared[STRIDE*k+3];
	}
      }
    }

    // Write output
    target_idx = i*total_target_width + target_offset + f*width + j;
    target[target_idx] = Pvalue;
  }
}

__global__ void gradient_reduce(const {{ data_type }} *df_weights,
				{{ data_type }} *df_weights_sum, 
				const unsigned int n_filters,
				const unsigned int filter_width, 
				const unsigned int n_elements) {

  /* 
     Reduction operation necessary to complete the gradient computation
  */
    
  const unsigned int tid = threadIdx.x;
  const unsigned int filter_elements = STRIDE*filter_width;
  const unsigned int df_weights_idx = blockIdx.x*filter_elements*n_elements+
    blockIdx.y*n_elements+threadIdx.x;
    
  extern __shared__ {{ data_type }} sdata[];
    
  sdata[tid] = (tid<n_elements) ? df_weights[df_weights_idx] : 0;
  if (tid+blockDim.x < n_elements)
    sdata[tid] += df_weights[df_weights_idx+blockDim.x];
  __syncthreads();
    
  for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
    if (tid < s) {
      sdata[tid] += sdata[tid+s];
    }
    __syncthreads();
  }
    
  if (tid==0) {        
    const unsigned int df_weights_sum_idx = blockIdx.x*filter_elements+blockIdx.y;
    df_weights_sum[df_weights_sum_idx] = sdata[0];
  }
}

__global__ void convolve_sequence_gradient(const nucleotide_t *input, 
					   const {{ data_type }} *df_output,
					   {{ data_type }} *df_weights, 
					   const unsigned int input_offset,
					   const unsigned int df_output_offset,
					   const unsigned int total_input_width,
					   const unsigned int total_df_output_width,
					   const unsigned int width,
					   const unsigned int height, 
					   const unsigned int filter_width,
					   const unsigned int n_filters) {

  /*
    Compute the gradient of the convolution operation with respect to the filter weights
    
    *input : pointer to the input sequence
    *df_output : pointer to the incoming gradient from the next layer
    *df_weights : pointer to output array for gradient wrt filter weights
    input_offset : column offset into input array
    df_output_offset : column offset into df_output
    total_input_width : total_width/row stride of input array
    total_df_output_width : total_width/row stride of df_output
    width : input width
    height : number of input rows
    filter_width : width of filters
    n_filters : number of filters in filter bank

  */

  
  const unsigned int tx = threadIdx.x;
  const unsigned int f = blockIdx.y;
  const unsigned int lin_idx = blockIdx.x*blockDim.x+tx;
  const unsigned int row = lin_idx / width;
  const unsigned int column = lin_idx % width;
  const unsigned int input_idx = row*total_input_width + input_offset + column;
  const unsigned int column_start_block = (blockIdx.x*blockDim.x)%width; // Column of first thread in block
  const unsigned int row_start_block = (blockIdx.x*blockDim.x)/width; // Row of first thread in block
  const unsigned int len_input = height*width;

  unsigned int df_weights_idx, output_idx, shared_idx, df_output_shift;
  int halo_idx;

  // Define dynamically sized shared memory
  const unsigned int halo_width = filter_width - 1;
  const unsigned int shared_width = halo_width + blockDim.x;
  extern __shared__ {{ data_type }} sdata[];
  {{ data_type }} *df_output_shared = sdata;
  {{ data_type }} *df_weights_reduce = df_output_shared + shared_width;

  // Load input element
  const nucleotide_t input_element = 
    (lin_idx < len_input) ? input[input_idx] : DNA_N;

  // Load halo elements on the left into shared memory
  if (tx < halo_width) {
    output_idx = row_start_block*total_df_output_width +
      df_output_offset + f*width + column_start_block - halo_width + tx;
    shared_idx = tx;
    halo_idx = column_start_block - halo_width + tx;
    df_output_shared[shared_idx] = 
      (halo_idx < 0) ? 0. : df_output[output_idx];
  }

  // Load remaining shared memory elements
  if (tx < blockDim.x) {
    output_idx = row*total_df_output_width + df_output_offset + f*width + column;
    df_output_shared[tx+halo_width] = 
      (column < width && row < height) ?
      df_output[output_idx] : 0.;
  }
  
  __syncthreads();

  // Compute gradients
  for (unsigned int k=0; k<filter_width; k++) {
    df_output_shift = halo_width-k;

    if (column >= df_output_shift) {
      df_weights_reduce[STRIDE*tx] = (CHECK_NT(input_element, DNA_A)) ?
    	df_output_shared[tx+k] : 0.;

      df_weights_reduce[STRIDE*tx+1] = (CHECK_NT(input_element, DNA_C)) ?
    	df_output_shared[tx+k] : 0.;

      df_weights_reduce[STRIDE*tx+2] = (CHECK_NT(input_element, DNA_G)) ?
    	df_output_shared[tx+k] : 0.;

      df_weights_reduce[STRIDE*tx+3] = (CHECK_NT(input_element, DNA_T)) ?
    	df_output_shared[tx+k] : 0.;

      if (CHECK_NT(input_element, DNA_R)) {
	df_weights_reduce[STRIDE*tx] = .5 * df_output_shared[tx+k];
	df_weights_reduce[STRIDE*tx+2] = .5 * df_output_shared[tx+k];
      }
    
      if (CHECK_NT(input_element, DNA_Y)) {
	df_weights_reduce[STRIDE*tx+1] = .5 * df_output_shared[tx+k];
	df_weights_reduce[STRIDE*tx+3] = .5 * df_output_shared[tx+k];
      }

    } else {
      df_weights_reduce[STRIDE*tx] = 0.;
      df_weights_reduce[STRIDE*tx+1] = 0.;
      df_weights_reduce[STRIDE*tx+2] = 0.;
      df_weights_reduce[STRIDE*tx+3] = 0.;
    }

    __syncthreads();

    // Stage 1 reduction
    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
      if (tx<s) {
	df_weights_reduce[STRIDE*tx] += df_weights_reduce[STRIDE*(tx+s)];
	df_weights_reduce[STRIDE*tx+1] += df_weights_reduce[STRIDE*(tx+s)+1];
	df_weights_reduce[STRIDE*tx+2] += df_weights_reduce[STRIDE*(tx+s)+2];
	df_weights_reduce[STRIDE*tx+3] += df_weights_reduce[STRIDE*(tx+s)+3];
      }
      __syncthreads();
    }
    
    // Write output
    if (tx<STRIDE) {
      df_weights_idx =
	f * STRIDE * filter_width * gridDim.x +
	(tx + STRIDE * df_output_shift) * gridDim.x +
	blockIdx.x;
      df_weights[df_weights_idx] = df_weights_reduce[tx];
    }
  }
}

__global__ void max_pool(const {{ data_type }} *mat,
			 {{ data_type }} *target, 
			 unsigned int *argmax,
			 const unsigned int input_offset,
			 const unsigned int height,
			 const unsigned int total_width,
			 const unsigned int width,
			 const unsigned int pooled_offset,
			 const unsigned int total_width_pooled,
			 const unsigned int pool_size) {

  /* Perform 1D max-pooling on all rows of a matrix
   */
    
  const unsigned int tx = threadIdx.x;
  const unsigned int i = blockIdx.y;
  const unsigned int j = blockIdx.x*pool_size+tx;
  const unsigned int f = blockIdx.z;
  const unsigned int mat_idx = i*total_width + input_offset + f*width + j;
  const unsigned int width_pooled = CEILING(({{ data_type }}) width / pool_size);
    
  extern __shared__ {{ data_type }} sdata[];
  {{ data_type }} *max_shared = sdata;
  unsigned int *argmax_shared = (unsigned int*) (max_shared + blockDim.x);
    
  max_shared[tx] = (i < height && j < width && tx < pool_size) ? mat[mat_idx] : -FLT_MAX;
  argmax_shared[tx] = (i < height && j < width && tx < pool_size) ? j : UINT_MAX;
  __syncthreads();
    
  for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
    if (tx<s && sdata[tx+s] > sdata[tx]) {
      max_shared[tx] = max_shared[tx+s];
      argmax_shared[tx] = argmax_shared[tx+s];
    }
    __syncthreads();
  }
    
  if (tx==0) {
    const unsigned int target_idx = i*total_width_pooled +
      pooled_offset + f*width_pooled + blockIdx.x;
    target[target_idx] = max_shared[0];
    argmax[target_idx] = argmax_shared[0];
  }
}

__global__ void max_pool_gradient(
				  const unsigned int *argmax,
				  const {{ data_type }} *df_output,
				  {{ data_type }} *df_input,
				  const unsigned int input_offset,
				  const unsigned int height,
				  const unsigned int total_width,
				  const unsigned int width,
				  const unsigned int pooled_offset,
				  const unsigned int total_width_pooled,
				  const unsigned int width_pooled) {

  /* Gradient of max-pooling operation
   */
    
  const unsigned int tx = threadIdx.x;
  const unsigned int bx = blockIdx.x;
  const unsigned int f = blockIdx.y;
  const unsigned int row = blockIdx.z;
  const unsigned int column = bx*blockDim.x+tx;
    
  const unsigned int pooled_idx = row*total_width_pooled +
    pooled_offset + f*width_pooled + bx;
  const unsigned int max_idx = argmax[pooled_idx];
  const {{ data_type }} df_output_element = df_output[pooled_idx];

  if (column < width) {
    df_input[row*total_width + input_offset +
	     f*width + column] =
      (column == max_idx) ? df_output_element : 0.;
  }
}

__global__ void sum_pool(const {{ data_type }} *mat,
			 {{ data_type }} *target, 
			 const unsigned int input_offset,
			 const unsigned int height,
			 const unsigned int total_width,
			 const unsigned int width,
			 const unsigned int pooled_offset,
			 const unsigned int total_width_pooled,
			 const unsigned int pool_size) {

  /* Perform 1D sum-pooling on all rows of a matrix
   */
    
  const unsigned int tx = threadIdx.x;
  const unsigned int i = blockIdx.y;
  const unsigned int j = blockIdx.x*pool_size+tx;
  const unsigned int f = blockIdx.z;
  const unsigned int mat_idx = i*total_width + input_offset + f*width + j;
  const unsigned int width_pooled = CEILING(({{ data_type }}) width / pool_size);
    
  extern __shared__ {{ data_type }} sdata[];
  {{ data_type }} *sum_shared = sdata;
    
  sum_shared[tx] = (i < height && j < width && tx < pool_size) ? mat[mat_idx] : 0;
  __syncthreads();
    
  for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
    if (tx<s) {
      sum_shared[tx] += sum_shared[tx+s];
    }
    __syncthreads();
  }
    
  if (tx==0) {
    const unsigned int target_idx = i*total_width_pooled +
      pooled_offset + f*width_pooled + blockIdx.x;
    target[target_idx] = sum_shared[0];
  }
}

__global__ void sum_pool_gradient(
				  const {{ data_type }} *df_output,
				  {{ data_type }} *df_input,
				  const unsigned int input_offset,
				  const unsigned int height,
				  const unsigned int total_width,
				  const unsigned int width,
				  const unsigned int pooled_offset,
				  const unsigned int total_width_pooled,
				  const unsigned int width_pooled) {

  /* Gradient of sum-pooling operation
   */
    
  const unsigned int tx = threadIdx.x;
  const unsigned int bx = blockIdx.x;
  const unsigned int f = blockIdx.y;
  const unsigned int row = blockIdx.z;
  const unsigned int column = bx*blockDim.x+tx;
    
  const unsigned int pooled_idx = row*total_width_pooled +
    pooled_offset + f*width_pooled + bx;
  const {{ data_type }} df_output_element = df_output[pooled_idx];

  if (column < width) {
    df_input[row*total_width + input_offset +
	     f*width + column] = df_output_element;
  }
}

__global__ void fully_connected_layer(const nucleotide_t *input,
				      {{ data_type }} *target, 
				      const {{ data_type }} *filter,
				      const {{ data_type }} *bias,
				      const unsigned int input_offset,
				      const unsigned int width,
				      const unsigned int total_width,
				      const unsigned int height, 
				      const unsigned int total_target_width,
				      const unsigned int target_offset,
				      const unsigned int n_filters) {

  /* This is a simple variation of the convolution operation where the
     filter is the same size as the input. 
  */

  const unsigned int f = blockIdx.z;
  const unsigned int tx = threadIdx.x;
  const unsigned int dimy = blockDim.y;
  const unsigned int i = blockIdx.x*blockDim.x+threadIdx.x; // Row
  const unsigned int j = threadIdx.y; // Column
  const unsigned int filter_elements = STRIDE*width; // Actual number of elements in filter
  const nucleotide_t input_element = (i < height && j < width) ? 
    input[i*total_width+input_offset+j] : 0. ;
    
  extern __shared__ {{ data_type }} sdata[];
  {{ data_type }} *filter_shared = sdata; // size: filter_elements
  {{ data_type }} *output_shared = sdata + filter_elements; // size: blockDim.x * width

  // Load filter elements into shared memory
  const unsigned int tid = threadIdx.x*blockDim.y+threadIdx.y;
  if (tid < filter_elements)
    filter_shared[tid] = filter[f*filter_elements+tid];
  __syncthreads();

  const {{ data_type }} filter_A = filter_shared[j*STRIDE];
  const {{ data_type }} filter_C = filter_shared[j*STRIDE+1];
  const {{ data_type }} filter_G = filter_shared[j*STRIDE+2];
  const {{ data_type }} filter_T = filter_shared[j*STRIDE+3];

  // Compute output
  if (i < height && j < width) {
    {{ data_type }} Pvalue;
    if (CHECK_NT(input_element, DNA_A))
      Pvalue = filter_A;

    if (CHECK_NT(input_element, DNA_C))
      Pvalue = filter_C;

    if (CHECK_NT(input_element, DNA_G))
      Pvalue = filter_G;

    if (CHECK_NT(input_element, DNA_T))
      Pvalue = filter_T;

    if (CHECK_NT(input_element, DNA_R))
      Pvalue = .5 * (filter_A + filter_G);

    if (CHECK_NT(input_element, DNA_Y))
      Pvalue = .5 * (filter_C + filter_T);

    if (j == 0)
      Pvalue += bias[f];

    // Write output to shared memory
    output_shared[tx*dimy+j] = Pvalue;
  } else {
    output_shared[tx*dimy+j] = 0.;
  }
  __syncthreads();

  // Sum up all the filter elements
  for (unsigned int s=blockDim.y/2; s>0; s>>=1) {
    if (j < s) {
      output_shared[tx*dimy+j] += output_shared[tx*dimy+j+s];
    }
    __syncthreads();
  }

  // Write final output
  if (i < height and j == 0) {
    const unsigned int target_idx = i*total_target_width+target_offset+f;
    target[target_idx] = output_shared[tx*dimy];
  }
}

__global__ void fully_connected_layer_gradient(const nucleotide_t *input,
					       const {{ data_type }} *df_output,
					       {{ data_type }} *df_weights,
					       const unsigned int input_offset,
					       const unsigned int df_output_offset,
					       const unsigned int total_input_width,
					       const unsigned int total_df_output_width,
					       const unsigned int width,
					       const unsigned int height,
					       const unsigned int n_filters) {

  const unsigned int tx = threadIdx.x;
  const unsigned int i = blockDim.x*blockIdx.x+tx; // Row
  const unsigned int j = threadIdx.y; // Column
  const unsigned int f = threadIdx.z; // Filter
  const unsigned int input_idx = i*total_input_width+input_offset+j;
  
  const nucleotide_t input_element = input[input_idx];

  extern __shared__ {{ data_type }} sdata[];
  {{ data_type }} *df_output_shared = sdata; // size: blockDim.x
  {{ data_type }} *df_weights_shared = df_output_shared + blockDim.x; // size: blockDim.x * width * STRIDE

  // Load df_output into shared memory
  if (j == 0) {
    const unsigned int df_output_idx = i*total_df_output_width+df_output_offset+f;
    df_output_shared[tx] = df_output[df_output_idx];
  }
  __syncthreads();

  const {{ data_type }} df_output_element = df_output_shared[tx];
  
  // Compute gradient
  const unsigned int df_weights_shared_idx = tx*STRIDE*width+j*STRIDE;
  if (i < height && j < width) {
    // DNA_A
    df_weights_shared[df_weights_shared_idx] = CHECK_NT(input_element, DNA_A) ?
      df_output_element : 0;
    // DNA_C
    df_weights_shared[df_weights_shared_idx+1] = CHECK_NT(input_element, DNA_C) ?
      df_output_element : 0;
    // DNA_G
    df_weights_shared[df_weights_shared_idx+2] = CHECK_NT(input_element, DNA_G) ?
      df_output_element : 0;
    // DNA_T
    df_weights_shared[df_weights_shared_idx+3] = CHECK_NT(input_element, DNA_T) ?
      df_output_element : 0;

    if (CHECK_NT(input_element, DNA_R)) {
      df_weights_shared[df_weights_shared_idx] = .5 * df_output_element;
      df_weights_shared[df_weights_shared_idx+2] = .5 * df_output_element;
    }

    if (CHECK_NT(input_element, DNA_Y)) {
      df_weights_shared[df_weights_shared_idx+1] = .5 * df_output_element;
      df_weights_shared[df_weights_shared_idx+3] = .5 * df_output_element;
    }
  } else {
    df_weights_shared[df_weights_shared_idx] = 0.;
    df_weights_shared[df_weights_shared_idx+1] = 0.;
    df_weights_shared[df_weights_shared_idx+2] = 0.;
    df_weights_shared[df_weights_shared_idx+2] = 0.;
  }
  __syncthreads();

  // Stage 1 reduction
  unsigned int df_weights_shared_idx_next;
  for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
    if (tx < s) {
      df_weights_shared_idx_next = (tx+s)*STRIDE*width+j*STRIDE;
      df_weights_shared[df_weights_shared_idx] += df_weights_shared[df_weights_shared_idx_next];
      df_weights_shared[df_weights_shared_idx+1] += df_weights_shared[df_weights_shared_idx_next+1];
      df_weights_shared[df_weights_shared_idx+2] += df_weights_shared[df_weights_shared_idx_next+2];
      df_weights_shared[df_weights_shared_idx+3] += df_weights_shared[df_weights_shared_idx_next+3];
    }
    __syncthreads();
  }
  
  // Write output
  if (tx==0) {
    const unsigned int df_weights_idx = blockIdx.x*STRIDE*width+j*STRIDE;
    df_weights[df_weights_idx] = df_weights_shared[j*STRIDE];
    df_weights[df_weights_idx+1] = df_weights_shared[j*STRIDE+1];
    df_weights[df_weights_idx+2] = df_weights_shared[j*STRIDE+2];
    df_weights[df_weights_idx+3] = df_weights_shared[j*STRIDE+3];
  }
}
